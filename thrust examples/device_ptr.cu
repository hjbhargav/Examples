#include "hip/hip_runtime.h"
#include <thrust/transform_reduce.h>
#include <thrust/device_vector.h>
#include <thrust/pair.h>
#include <thrust/random.h>
#include <thrust/extrema.h>

// This example shows how to compute a bounding box
// for a set of points in two dimensions.

struct point2d
{
  float x, y;
  
  __host__ __device__
  point2d() : x(0), y(0) {}
  
  __host__ __device__
  point2d(float _x, float _y) : x(_x), y(_y) {}
};

// bounding box type
struct bbox
{
  // construct an empty box
  __host__ __device__
  bbox() {}

  // construct a box from a single point
  __host__ __device__
  bbox(const point2d &point)
    : lower_left(point), upper_right(point)
  {}

  // construct a box from a pair of points
  __host__ __device__
  bbox(const point2d &ll, const point2d &ur)
    : lower_left(ll), upper_right(ur)
  {}

  point2d lower_left, upper_right;
};

// reduce a pair of bounding boxes (a,b) to a bounding box containing a and b
struct bbox_reduction : public thrust::binary_function<bbox,bbox,bbox>
{
  __host__ __device__
  bbox operator()(bbox a, bbox b)
  {
    // lower left corner
    point2d ll(thrust::min(a.lower_left.x, b.lower_left.x), thrust::min(a.lower_left.y, b.lower_left.y));
    
    // upper right corner
    point2d ur(thrust::max(a.upper_right.x, b.upper_right.x), thrust::max(a.up